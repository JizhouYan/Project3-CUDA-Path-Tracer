#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
namespace Thrust {


	using StreamCompaction::Common::PerformanceTimer;
	PerformanceTimer& timer()
	{
		static PerformanceTimer timer;
		return timer;
	}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
    // use `thrust::exclusive_scan`
    // example: for device_vectors dv_in and dv_out:
    // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

	thrust::host_vector<int> host_input(idata, idata + n);
	thrust::device_vector<int> dev_input = host_input;

	//thrust::host_vector<int> host_output(odata, odata + n);
	thrust::device_vector<int> dev_output(odata, odata + n);

	// what happened during thrust? GPU timer malfunctioning
	timer().startGpuTimer();
	// call
	thrust::exclusive_scan(dev_input.begin(), dev_input.end(), dev_output.begin());

	timer().endGpuTimer();


	thrust::copy(dev_output.begin(), dev_output.end(), odata);
}

}
}
